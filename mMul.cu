#include "hip/hip_runtime.h"
#include <stdio.h>
#include "matrixMul.h"

///////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: P = M * N
///////////////////////////////////////////////////////////////////////////////

__global__ void
matrixMulKernelGlobal( float* Md, float* Nd, float* Pd, int width)
{

    // Thread index
    int k;
    float Psub = 0.0f;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < width && Col < width) {
    	for (k = 0; k < width; ++k) {
	        Psub += Md[Row * width + k] * Nd[k * width + Col];
	    }

	    Pd[Row * width + Col] = Psub;
    }
}

__global__ void
matrixMulKernelShared( float* Md, float* Nd, float* Pd, int width)
{
    __shared__ float Rmem[ROW_SIZE][K_SIZE];
    __shared__ float Cmem[K_SIZE][COLUMN_SIZE];

    // Thread index
    int k, r, c, K, R, C, K_Block;
    // int tid = threadIdx.x * blockDim.y + threadIdx.y;
    float Psub;
    int Row = blockIdx.y * ROW_SIZE + threadIdx.y;
    int Col = blockIdx.x * COLUMN_SIZE + threadIdx.x;

    // initialize submatrix to 0
    
    for (r=0; r < ROW_SIZE; r += THREAD_BLOCK_0) {
        R = r + Row;
        for (c=0; c < COLUMN_SIZE; c += THREAD_BLOCK_1) {
            C = c + Col;
            // if (R < width && C < width) {

                Pd[R * width + C] = 0.0f;
            // }
        }
    }


    for (K_Block=0; K_Block < width; K_Block += K_SIZE) {

        // copy in C submatrix
        for (k=0; k < K_SIZE; k += THREAD_BLOCK_0) {
            K = k + K_Block + threadIdx.y;
            for (c=0; c < COLUMN_SIZE; c += THREAD_BLOCK_1) {
                C = c + Col;
                // if (K < width && C < width)
                {
                    Cmem[k + threadIdx.y][c + threadIdx.x] = Nd[K * width + C];
                    // printf("C[%i,%i]=%.3f\n", c+threadIdx.x, k + threadIdx.y, Cmem[c+threadIdx.x][k + threadIdx.y]);
                }
            }
        }
        // if (Row + Col == 0)printf("++++++++++++++++++++++++++\n");

        // copy in R submatrix
        for (r=0; r < ROW_SIZE; r += THREAD_BLOCK_0) {
            R = r + Row;
            for (k=0; k < K_SIZE; k += THREAD_BLOCK_1) {
                K = k + K_Block + threadIdx.x;
                // if (K < width && R < width) 
                {
                    Rmem[r + threadIdx.y][k + threadIdx.x] = Md[R * width + K];
                    // printf("R[%i,%i]=%.3f\n", r+threadIdx.y, k + threadIdx.x, Rmem[r+threadIdx.y][k + threadIdx.x]);
                }
            }
        }
        // if (Row + Col == 0)printf("--------------------------\n");

        // ensure data read in before use
        __syncthreads();

        for (c=0; c < COLUMN_SIZE; c += THREAD_BLOCK_1) {
            C = c + Col;
            for (r=0; r < ROW_SIZE; r += THREAD_BLOCK_0) {
                R = r + Row;
                // if (R < width && C < width) 
                {

                    Psub = 0.0f;
                    for (k=0; k < K_SIZE /*&& k < width - K_Block*/; k++) {
                        
                        Psub += Rmem[r + threadIdx.y][k] * Cmem[k][c + threadIdx.x];
                        // printf("C[%i,%i]=%.3f\n", c + threadIdx.x, k, Cmem[c + threadIdx.x][k]);
                        // printf("R[%i,%i]=%.3f\n", r + threadIdx.y, k, Rmem[r + threadIdx.y][k]);
                    }

                    // printf("O[%i,%i]=%.3f\n", R, C, Psub);

                    Pd[R * width + C] += Psub;
                }
            }
        }
        // ensure data used before overwritten
        __syncthreads();
    }
}