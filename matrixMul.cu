#include "hip/hip_runtime.h"
//
// Compile: nvcc matrixMul.cu matrixMul_gold.cpp -o mMul
// Use: mMul
//
#include <stdio.h>
#include "matrixMul.h"

// includes, kernels
#include "mMul.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int, int, float);

extern "C"
void computeGold( float*, const float*, const float*, unsigned int w);

void printMatrix(float *M, int width) {
    int i, j;
    for (i=0;i<width;i++){
        for (j=0;j<width;j++){
            printf("%.3f ",M[i*width+j]);
        }
        printf("\n");
    }
    printf("\n");
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // set seed for rand()
    srand(2006);

    if (argc != 2) {
        printf("usage: ./mMul [size of matrix]\n");
        exit(1);
    }

    int width = atoi(argv[1]); 

    // allocate host memory for matrices M and N
    unsigned int size_M = width * width;
    unsigned int mem_size_M = sizeof(float) * size_M;
    float* h_M = (float*)malloc(mem_size_M);
    unsigned int size_N = width * width;
    unsigned int mem_size_N = sizeof(float) * size_N;
    float* h_N = (float*)malloc(mem_size_N);

    // initialize host memory
    randomInit(h_M, size_M);
    randomInit(h_N, size_N);

    // allocate device memory
    float* d_M;
    hipMalloc((void**) &d_M, mem_size_M);
    float* d_N;
    hipMalloc((void**) &d_N, mem_size_N);

    hipEventRecord(start, 0);

    // copy host memory to device
    hipMemcpy(d_M, h_M, mem_size_M,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, mem_size_N,
                              hipMemcpyHostToDevice);

    // allocate device memory for result
    unsigned int size_P = width * width;
    unsigned int mem_size_P = sizeof(float) * size_P;
    float* d_P;
    hipMalloc((void**) &d_P, mem_size_P);

    // allocate host memory for the result
    float* h_P = (float*) malloc(mem_size_P);

    // printMatrix(h_N,width);
    // printMatrix(h_M,width);
    

    // setup execution parameters
    dim3 blocks(ceil(width/(double)ROW_SIZE), ceil(width/(double)COLUMN_SIZE), 1);
    dim3 threads(THREAD_BLOCK_0, THREAD_BLOCK_1, 1);

    // kernel warmup
    // matrixMulKernelGlobal<<< blocks, threads >>>(d_M, d_N, d_P, width);
    matrixMulKernelShared<<< blocks, threads >>>(d_M, d_N, d_P, width);
    hipDeviceSynchronize();
    
    // copy result from device to host
    hipMemcpy(h_P, d_P, mem_size_P,
                              hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Elapsed time = %f ms\n", time);

    // compute reference solution
    float* reference = (float*)malloc(mem_size_P);
    computeGold(reference, h_M, h_N, width);

    // check result
    printDiff(reference, h_P, width, width, 100, 1.0e-5f);

    // clean up memory
    free(h_M);
    free(h_N);
    free(h_P);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;
    for (j = 0; j < height; j++) 
    {
        if (error_count < iListLength)
        {
            printf("\n  Row %d:\n", j);
        }
        for (i = 0; i < width; i++) 
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]) / data1[k];
            if (fDiff > fListTol) 
            {                
                if (error_count < iListLength)
                {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    printf(" \n  Total Errors = %d\n\n", error_count);
}

